/*

FILENAME:  gpu_hello.cu


Copy the string "hello, world" from CPU to GPU and back
using common CUDA methods.

Naming convention of the GPU world:
H_  -  host (CPU)
D_  -  device (GPU)

*/


#include <hip/hip_runtime.h>    /* GPU library                           */
#include <stdio.h>   /* printf()                              */


/* Forward Reference                                          */
__global__ void HelloWorld (char*,char*);

int main(int argc, char** argv) {

    /* 1) The host initializes an array.                      */
    /*    - define source message and target array.           */
    /*    - allocate memory on the host.                      */
    char H_str1[] = "hello, world";
    char H_str2[] = "XXXXXXXXXXXX";

    /* Set device based on input from command line            */
    if (argc > 1) {
        if (hipSetDevice(atoi(argv[1])) != hipSuccess) {
            int num_devices;
            hipGetDeviceCount(&num_devices);
            fprintf(stderr, "Error initializing device %s,\
 device value must be 0-%d\n", argv[1], (num_devices-1));
            return 0;
        }
    } else {
        fprintf(stderr, "No GPU specified, using first GPU");
        if (hipSetDevice(0) != hipSuccess) {
            int num_devices;
            hipGetDeviceCount(&num_devices);
            fprintf(stderr, "Error initializing device 0,\
 device value must be 0-%d\n", (num_devices-1));
            return 0;
        }
    }

    /* Allocate memory on the GPU device.                     */
    char *D_str1, *D_str2;
    size_t size = sizeof(H_str1);     /* 13 characters        */
    hipMalloc((void**)&D_str1, size);
    hipMalloc((void**)&D_str2, size);

    /* 2) Copy array from host memory to GPU memory.          */
    hipMemcpy(D_str1, H_str1, size, hipMemcpyHostToDevice);

    /* Set the grid and block sizes.                          */
    dim3 dimGrid(1);
    dim3 dimBlock(size);     /* one thread per character      */

    /* 3) GPU operates on the array.                          */
    /*    - invoke the kernel.                                */
    HelloWorld<<< dimGrid, dimBlock >>>(D_str1,D_str2);

    /* 4) Copy array from GPU memory to host memory.          */
    hipMemcpy(H_str2, D_str2, size, hipMemcpyDeviceToHost);

    /* Free up the allocated memory on the GPU.               */
    hipFree(D_str1);
    hipFree(D_str2);

    /* Display result of the copy.                            */
    printf("%s\n", H_str2);

    return 0;
}

/* Device Kernel                                              */
/* On the GPU, perform some computation (copy).               */
__global__ void HelloWorld(char* str1, char* str2) {
    /* Determine thread ID.                                   */
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    /* Copy one element of the string.                        */
    str2[idx] = str1[idx];
}
